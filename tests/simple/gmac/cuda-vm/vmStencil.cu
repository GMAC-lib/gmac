#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmac/hip/hip_runtime.h>

#include "utils.h"
#include "debug.h"
#include "barrier.h"

#include "vmStencilCommon.cu"


int main(int argc, char *argv[])
{
	setParam<size_t>(&dimRealElems, dimRealElemsStr, dimRealElemsDefault);

    if (dimRealElems % 32 != 0) {
        fprintf(stderr, "Error: wrong dimension %zd\n", dimRealElems);
        abort();
    }

    dimElems = dimRealElems + 2 * STENCIL;

    JobDescriptor * descriptor = new JobDescriptor();
    descriptor->gpus  = 1;
    descriptor->gpuId = 1;

    descriptor->prev = NULL;
    descriptor->next = NULL;

    descriptor->dimRealElems = dimRealElems;
    descriptor->dimElems     = dimElems;
    descriptor->slices       = dimRealElems;

    do_stencil((void *) descriptor);

    delete descriptor;
}
