#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gmac/hip/hip_runtime.h>

#include "utils.h"

__global__ void kernelFill(unsigned *A, unsigned off, size_t size)
{
    unsigned localIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx = localIdx + off;

    if (idx >= size) return;
    A[localIdx] = idx;
}

int main(int argc, char *argv[])
{
    const unsigned totalSize = 8 * 1024 * 1024;
    gmactime_t s, t;

    for (unsigned currentSize = totalSize; currentSize > 32; currentSize /= 2) {
        assert(totalSize % currentSize == 0);
        fprintf(stderr,"Size: %u\n", currentSize);
        size_t nObjects = totalSize / currentSize;
        unsigned **objects = (unsigned **) malloc(nObjects * sizeof(int *));
        assert(objects != NULL);

        getTime(&s);
        for(size_t i = 0; i < nObjects; i++) {
            assert(gmacMalloc((void **)&objects[i], currentSize * sizeof(int)) == gmacSuccess);
        }
        getTime(&t);
        printTime(&s, &t, "Alloc: ", "\n");

        getTime(&s);
        unsigned off = 0;
        dim3 Db(currentSize > 256? 256: currentSize);
        dim3 Dg(currentSize / Db.x);
        if (currentSize > 256 && currentSize % 256 != 0) Dg.x++;

        for(size_t i = 0; i < nObjects; i++) {
            kernelFill<<<Dg, Db>>>(gmacPtr(objects[i]), off, totalSize);
            off += currentSize;
        }
        gmacThreadSynchronize();
        getTime(&t);
        printTime(&s, &t, "Run: ", "\n");

        getTime(&s);
        off = 0;
        for(size_t i = 0; i < nObjects; i++) {
            for(size_t j = 0; j < currentSize; j++) {
                size_t idx = off + j;
                assert(objects[i][j] == idx);
            }
            off += currentSize;
        }
        getTime(&t);
        printTime(&s, &t, "Check: ", "\n");

        getTime(&s);
        for(size_t i = 0; i < nObjects; i++) {
            gmacFree(objects[i]);
        }
        free(objects);
        getTime(&t);
        printTime(&s, &t, "Free: ", "\n");
    }

    return 0;
}
