#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <gmac/cuda>

#include "utils.h"
#include "debug.h"


const char *vecSizeStr = "GMAC_VECSIZE";
const size_t vecSizeDefault = 16 * 1024 * 1024;

size_t vecSize = 0;
const size_t blockSize = 512;

__global__ void vecAdd(float *c, float *a, float *b, size_t size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= size) return;

    c[i] = a[i] + b[i];
}

void init(float *ptr, unsigned s, float v)
{
    for(unsigned i = 0; i < s; i++) {
        ptr[i] = v;
    }
}


int main(int argc, char *argv[])
{
	float *a, *b, *c;
	gmactime_t s, t, begin, end;

	setParam<size_t>(&vecSize, vecSizeStr, vecSizeDefault);

    getTime(&s);
    // Alloc input data
    a = new (gmac::allocator) float[vecSize];
    b = new (gmac::allocator) float[vecSize];
    // Alloc output data
    c = new (gmac::allocator) float[vecSize];

    assert(a != NULL);
    assert(b != NULL);
    assert(c != NULL);
    getTime(&t);
    printTime(&s, &t, "Alloc: ", "\n");

    float sum = 0.f;

    // Init input data
    getTime(&s);
    begin = s;
    randInitMax(a, 10.f, vecSize);
    randInitMax(b, 10.f, vecSize);
    getTime(&t);
    printTime(&s, &t, "Init: ", "\n");

    for(unsigned i = 0; i < vecSize; i++) {
        sum += a[i] + b[i];
    }
    
    // Call the kernel
    getTime(&s);
    dim3 Db(blockSize);
    dim3 Dg((unsigned long)vecSize / blockSize);
    if(vecSize % blockSize) Dg.x++;
    vecAdd<<<Dg, Db>>>(gmac::ptr(c), gmac::ptr(a), gmac::ptr(b), vecSize);
    assert(gmac::threadSynchronize() == gmacSuccess);
    getTime(&t);
    printTime(&s, &t, "Run: ", "\n");

    getTime(&s);
    float check = 0;
    for(unsigned i = 0; i < vecSize; i++) {
        check += c[i];
    }
    getTime(&t);
    end = t;
    printTime(&s, &t, "Check: ", "\n");

    getTime(&s);
    gmac::free(a);
    gmac::free(b);
    gmac::free(c);
    getTime(&t);
    printTime(&s, &t, "Free: ", "\n");
    printTime(&begin, &end, "Total: ", "\n");

    return sum != check;
}
