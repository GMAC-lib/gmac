#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#include <pthread.h>
#include <semaphore.h>

#include <gmac/hip/hip_runtime.h>

#include "utils.h"
#include "debug.h"

#include "gmacCompress.h"

const char *widthStr = "GMAC_WIDTH";
const char *heightStr = "GMAC_HEIGHT";
const char *framesStr = "GMAC_FRAMES";

const unsigned widthDefault = 128;
const unsigned heightDefault = 128;
const unsigned framesDefault = 32;

unsigned width = 0;
unsigned height = 0;
unsigned frames = 0;
const unsigned blockSize = 16;

static float *quant_in, *idct_in;

static pthread_t dct_id, quant_id, idct_id;
static gmac_sem_t quant_data, idct_data;
static gmac_sem_t quant_free, idct_free;

double timeDCTAlloc = 0.0;
double timeDCTInit  = 0.0;
double timeDCTCopy  = 0.0;
double timeDCTRun   = 0.0;
double timeDCTFree  = 0.0;

double timeQuantAlloc = 0.0;
double timeQuantCopy  = 0.0;
double timeQuantRun   = 0.0;
double timeQuantFree  = 0.0;

double timeIDCTAlloc = 0.0;
double timeIDCTRun   = 0.0;
double timeIDCTFree  = 0.0;

void __randInit(float *a, unsigned size)
{
	for(unsigned i = 0; i < size; i++) {
		a[i] = 10.0 * rand() / RAND_MAX;
	}
}

void *dct_thread(void *args)
{
	float *in, *out;
	gmacError_t ret;
    gmactime_t s, t;

    getTime(&s);
	ret = gmacMalloc((void **)&in, width * height * sizeof(float));
	assert(ret == gmacSuccess);
	ret = gmacMalloc((void **)&out, width * height * sizeof(float));
	assert(ret == gmacSuccess);
    getTime(&t);
    timeDCTAlloc += getTimeStamp(t) - getTimeStamp(s);

	dim3 Db(blockSize, blockSize);
	dim3 Dg(width / blockSize, height / blockSize);
	if(width % blockSize) Dg.x++;
	if(height % blockSize) Dg.y++;

	for(unsigned i = 0; i < frames; i++) {
        getTime(&s);
		__randInit(in, width * height);
        getTime(&t);
        timeDCTInit += getTimeStamp(t) - getTimeStamp(s);

        getTime(&s);
		dct<<<Dg, Db>>>(gmacPtr(out), gmacPtr(in), width, height);
		ret = gmacThreadSynchronize();
		assert(ret == gmacSuccess);
        getTime(&t);
        timeDCTRun += getTimeStamp(t) - getTimeStamp(s);

        getTime(&s);
		gmac_sem_wait(&quant_free, 1); /* Wait for quant to use its data */
		gmacMemcpy(quant_in, out, width * height * sizeof(float));
		gmac_sem_post(&quant_data, 1); /* Notify to Quant that data is ready */
        getTime(&t);
        timeDCTCopy += getTimeStamp(t) - getTimeStamp(s);
	}

    getTime(&s);
	gmacFree(in);
	gmacFree(out);
    getTime(&t);
    timeDCTFree += getTimeStamp(t) - getTimeStamp(s);

	return NULL;
}

void *quant_thread(void *args)
{
	float *out;
	gmacError_t ret;
    gmactime_t s, t;

    getTime(&s);
	ret = gmacMalloc((void **)&quant_in, width * height * sizeof(float));
	assert(ret == gmacSuccess);
	ret = gmacMalloc((void **)&out, width * height * sizeof(float));
	assert(ret == gmacSuccess);
    getTime(&t);
    timeQuantAlloc += getTimeStamp(t) - getTimeStamp(s);

	dim3 Db(blockSize, blockSize);
	dim3 Dg(width / blockSize, height / blockSize);
	if(width % blockSize) Dg.x++;
	if(height % blockSize) Dg.y++;

	gmac_sem_post(&quant_free, 1);

	for(unsigned i = 0; i < frames; i++) {
        getTime(&s);
		gmac_sem_wait(&quant_data, 1);	/* Wait for data to be processed */
		quant<<<Dg, Db>>>(gmacPtr(quant_in), gmacPtr(out), width, height, 1e-6);
		ret = gmacThreadSynchronize();
		assert(ret == gmacSuccess);
        getTime(&t);
        timeQuantRun += getTimeStamp(t) - getTimeStamp(s);
		
        getTime(&s);
		gmac_sem_wait(&idct_free, 1); /* Wait for IDCT to use its data */
		gmacMemcpy(idct_in, out, width * height * sizeof(float));
		gmac_sem_post(&quant_free, 1); /* Notify to DCT that Quant is waiting for data */
		gmac_sem_post(&idct_data, 1); /* Nodify to IDCT that data is ready */
        getTime(&t);
        timeQuantCopy += getTimeStamp(t) - getTimeStamp(s);
	}

    getTime(&s);
	gmacFree(quant_in);
	gmacFree(out);
    getTime(&t);
    timeQuantFree += getTimeStamp(t) - getTimeStamp(s);

	return NULL;
}

void *idct_thread(void *args)
{
	float *out;
	gmacError_t ret;
    gmactime_t s, t;

    getTime(&s);
	ret = gmacMalloc((void **)&idct_in, width * height * sizeof(float));
	assert(ret == gmacSuccess);
	ret = gmacMalloc((void **)&out, width * height * sizeof(float));
	assert(ret == gmacSuccess);
    getTime(&t);
    timeIDCTAlloc += getTimeStamp(t) - getTimeStamp(s);

	dim3 Db(blockSize, blockSize);
	dim3 Dg(width / blockSize, height / blockSize);
	if(width % blockSize) Dg.x++;
	if(height % blockSize) Dg.y++;

	gmac_sem_post(&idct_free, 1);

	for(unsigned i = 0; i < frames; i++) {
        getTime(&s);
		gmac_sem_wait(&idct_data, 1);
		idct<<<Dg, Db>>>(gmacPtr(idct_in), gmacPtr(out), width, height);
		ret = gmacThreadSynchronize();
		assert(ret == gmacSuccess);

		gmac_sem_post(&idct_free, 1);
        getTime(&t);
        timeIDCTRun += getTimeStamp(t) - getTimeStamp(s);
	}

    getTime(&s);
	gmacFree(idct_in);
	gmacFree(out);
    getTime(&t);
    timeIDCTFree += getTimeStamp(t) - getTimeStamp(s);

	return NULL;
}


int main(int argc, char *argv[])
{
	gmactime_t s,t;
	setParam<unsigned>(&width, widthStr, widthDefault);
	setParam<unsigned>(&height, heightStr, heightDefault);
	setParam<unsigned>(&frames, framesStr, framesDefault);

	gmac_sem_init(&quant_data, 0); 
	gmac_sem_init(&quant_free, 0); 
	gmac_sem_init(&idct_data,  0); 
	gmac_sem_init(&idct_free,  0); 

	srand(time(NULL));

	getTime(&s);

	pthread_create(&dct_id, NULL, dct_thread, NULL);
	pthread_create(&quant_id, NULL, quant_thread, NULL);
	pthread_create(&idct_id, NULL, idct_thread, NULL);

	pthread_join(dct_id, NULL);
	pthread_join(quant_id, NULL);
	pthread_join(idct_id, NULL);

	getTime(&t);

	fprintf(stdout, "DCT-Alloc: %f\n", timeDCTAlloc);
	fprintf(stdout, "DCT-Init: %f\n", timeDCTInit);
	fprintf(stdout, "DCT-Run: %f\n", timeDCTRun);
	fprintf(stdout, "DCT-Copy: %f\n", timeDCTCopy);
	fprintf(stdout, "DCT-Free: %f\n", timeDCTFree);

	fprintf(stdout, "Quant-Alloc: %f\n", timeQuantAlloc);
	fprintf(stdout, "Quant-Run: %f\n", timeQuantRun);
	fprintf(stdout, "Quant-Copy: %f\n", timeQuantCopy);
	fprintf(stdout, "Quant-Free: %f\n", timeQuantFree);

	fprintf(stdout, "IDCT-Alloc: %f\n", timeIDCTAlloc);
	fprintf(stdout, "IDCT-Run: %f\n", timeIDCTRun);
	fprintf(stdout, "IDCT-Free: %f\n", timeIDCTFree);

	printTime(&s, &t, "Total: ", "\n");

    return 0;
}
